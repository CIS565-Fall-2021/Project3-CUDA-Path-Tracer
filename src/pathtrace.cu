#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>

#include <cmath>
#include <cstdio>

#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "glm/gtx/norm.hpp"
#include "interactions.h"
#include "intersections.h"
#include "pathtrace.h"
#include "scene.h"
#include "sceneStructs.h"
#include "static_config.h"
#include "stream_compaction.h"
#include "utilities.h"

#define ERRORCHECK 1

#define FILENAME \
  (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (hipSuccess == err) {
    return;
  }

  fprintf(stderr, "CUDA error");
  if (file) {
    fprintf(stderr, " (%s:%d)", file, line);
  }
  fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
  getchar();
#endif
  exit(EXIT_FAILURE);
#endif
}

__host__ __device__ thrust::default_random_engine makeSeededRandomEngine(
    int iter, int index, int depth) {
  int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
  return thrust::default_random_engine(h);
}

// Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4 *pbo, glm::ivec2 resolution, int iter,
                               glm::vec3 *image) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index     = x + (y * resolution.x);
    glm::vec3 pix = image[index];

    glm::ivec3 color;
    color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
    color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
    color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

    // Each thread writes one pixel location in the texture (textel)
    pbo[index].w = 0;
    pbo[index].x = color.x;
    pbo[index].y = color.y;
    pbo[index].z = color.z;
  }
}

// Send Normal vector in GBuffer for visualization
__global__ void gbufferNormalToPBO(uchar4 *pbo, glm::ivec2 resolution,
                                   GBufferPixel *gBuffer) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index = x + (y * resolution.x);

    pbo[index].w = 0;
    pbo[index].x = fabs(gBuffer[index].normal.x * 255.0);
    pbo[index].y = fabs(gBuffer[index].normal.y * 255.0);
    pbo[index].z = fabs(gBuffer[index].normal.z * 255.0);
  }
}

// Send Position vector in GBuffer for visualization
__global__ void gbufferPositionToPBO(uchar4 *pbo, glm::ivec2 resolution,
                                     GBufferPixel *gBuffer,
                                     glm::vec3 scene_min_xyz,
                                     glm::vec3 scene_max_xyz) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index          = x + (y * resolution.x);
    glm::vec3 position = gBuffer[index].position;
    float hit          = (float)(gBuffer[index].t > 0);

    pbo[index].w = 0;
    pbo[index].x = 255.0 * hit * (position.x - scene_min_xyz.x) /
                   (scene_max_xyz.x - scene_min_xyz.x);
    pbo[index].y = 255.0 * hit * (position.y - scene_min_xyz.y) /
                   (scene_max_xyz.y - scene_min_xyz.y);
    pbo[index].z = 255.0 * hit * (position.z - scene_min_xyz.z) /
                   (scene_max_xyz.z - scene_min_xyz.z);
  }
}

// Send denoised weights for visualization
__global__ void gbufferWeightToPBO(uchar4 *pbo, glm::ivec2 resolution,
                                   float *weights) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int index    = x + (y * resolution.x);
    float weight = weights[index];

    pbo[index].w = 0;
    pbo[index].x = 255.0 * weight;
    pbo[index].y = 255.0 * weight;
    pbo[index].z = 255.0 * weight;
  }
}

static Scene *hst_scene                         = NULL;
static glm::vec3 *dev_image                     = NULL;
static Geom *dev_geoms                          = NULL;
static Material *dev_materials                  = NULL;
static PathSegment *dev_paths                   = NULL;
static ShadeableIntersection *dev_intersections = NULL;
static GBufferPixel *dev_gBuffer                = NULL;
static int *dev_materialIDs                     = NULL;
static int *dev_materialIDBuffers               = NULL;
static glm::vec3 *dev_image_buffer              = NULL;

// first-bounce intersection caching
#ifdef CACHE_INTERSECTIONS
static ShadeableIntersection *dev_intersections_cache = NULL;
static int *dev_materialIDs_cache                     = NULL;
#endif

// denoising parameters
static glm::ivec2 *dev_kernelOffset         = NULL;
static glm::vec3 *dev_image_denoised        = NULL;
static glm::vec3 *dev_image_denoised_buffer = NULL;
static float *dev_weights                   = NULL;  // for debug
static float *dev_posWeights                = NULL;  // for debug
static float *dev_norWeights                = NULL;  // for debug
static float *dev_colorWeights              = NULL;  // for debug

// 5x5 Gaussian kernel for image denoising
static const std::array<float, KERNEL_SIZE> kernel = {
    0.003765, 0.015019, 0.023792, 0.015019, 0.003765, 0.015019, 0.059912,
    0.094907, 0.059912, 0.015019, 0.023792, 0.094907, 0.150342, 0.094907,
    0.023792, 0.015019, 0.059912, 0.094907, 0.059912, 0.015019, 0.003765,
    0.015019, 0.023792, 0.015019, 0.003765};
__constant__ float cdev_kernel[KERNEL_SIZE];

// 5x5 offset for A-trous convolution
static std::vector<glm::ivec2> kernelOffset;

void pathtraceInit(Scene *scene) {
  hst_scene            = scene;
  const Camera &cam    = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

  hipMalloc(&dev_paths, ANTIALIAS_FACTOR * pixelcount * sizeof(PathSegment));

  hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
  hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
  hipMemcpy(dev_materials, scene->materials.data(),
             scene->materials.size() * sizeof(Material),
             hipMemcpyHostToDevice);

  hipMalloc(&dev_intersections,
             ANTIALIAS_FACTOR * pixelcount * sizeof(ShadeableIntersection));
  hipMemset(dev_intersections, 0,
             ANTIALIAS_FACTOR * pixelcount * sizeof(ShadeableIntersection));

  hipMalloc(&dev_gBuffer, pixelcount * sizeof(GBufferPixel));

  hipMalloc((void **)&dev_materialIDs,
             ANTIALIAS_FACTOR * pixelcount * sizeof(int));
  hipMalloc((void **)&dev_materialIDBuffers,
             ANTIALIAS_FACTOR * pixelcount * sizeof(int));
  hipMalloc(&dev_image_buffer, pixelcount * sizeof(glm::vec3));
#ifdef CACHE_INTERSECTIONS
  hipMalloc((void **)&dev_intersections_cache,
             ANTIALIAS_FACTOR * pixelcount * sizeof(ShadeableIntersection));
  hipMalloc((void **)&dev_materialIDs_cache,
             ANTIALIAS_FACTOR * pixelcount * sizeof(int));
#endif

  // ----- Denoising variables init -----
  // construct debugging buffers
  hipMalloc((void **)&dev_weights, pixelcount * sizeof(float));
  hipMalloc((void **)&dev_posWeights, pixelcount * sizeof(float));
  hipMalloc((void **)&dev_colorWeights, pixelcount * sizeof(float));
  hipMalloc((void **)&dev_norWeights, pixelcount * sizeof(float));
  checkCUDAError(
      "hipMalloc dev_weights, dev_posWeights, dev_colorWeights, "
      "dev_norWeights failed");
  // construct denoised image buffer
  hipMalloc((void **)&dev_image_denoised, pixelcount * sizeof(glm::vec3));
  hipMalloc((void **)&dev_image_denoised_buffer,
             pixelcount * sizeof(glm::vec3));
  checkCUDAError(
      "hipMalloc dev_image_denoised, dev_image_denoised_buffer failed");
  hipMemset(dev_image_denoised, 0, pixelcount * sizeof(glm::vec3));
  hipMemset(dev_image_denoised_buffer, 0, pixelcount * sizeof(glm::vec3));
  checkCUDAError("hipMemset dev_image_denoised failed");
  // construct kernel
  hipMemcpyToSymbol(HIP_SYMBOL(cdev_kernel), kernel.data(), KERNEL_SIZE * sizeof(float));
  checkCUDAError("hipMemcpyToSymbol to cdev_kernel failed");
  // construct convolution offsets
  for (int i = -KERNEL_WIDTH / 2; i <= KERNEL_WIDTH / 2; ++i) {
    for (int j = -KERNEL_WIDTH / 2; j <= KERNEL_WIDTH / 2; ++j) {
      kernelOffset.emplace_back(i, j);
    }
  }
  assert(kernelOffset.size() == KERNEL_SIZE);
  hipMalloc((void **)&dev_kernelOffset, KERNEL_SIZE * sizeof(glm::ivec2));
  checkCUDAError("hipMalloc dev_kernelOffset failed");
  hipMemcpy(dev_kernelOffset, kernelOffset.data(),
             KERNEL_SIZE * sizeof(glm::ivec2), hipMemcpyHostToDevice);
  checkCUDAError("hipMemcpy to dev_kernelOffset failed");

  checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
  hipFree(dev_image);  // no-op if dev_image is null
  hipFree(dev_paths);
  hipFree(dev_geoms);
  hipFree(dev_materials);
  hipFree(dev_intersections);
  hipFree(dev_gBuffer);
  hipFree(dev_materialIDs);
  hipFree(dev_materialIDBuffers);
  hipFree(dev_image_buffer);
#ifdef CACHE_INTERSECTIONS
  hipFree(dev_intersections_cache);
  hipFree(dev_materialIDs_cache);
#endif
  hipFree(dev_kernelOffset);
  hipFree(dev_image_denoised);
  hipFree(dev_image_denoised_buffer);
  hipFree(dev_weights);
  hipFree(dev_posWeights);
  hipFree(dev_norWeights);
  hipFree(dev_colorWeights);

  checkCUDAError("pathtraceFree");
}

#ifdef DEPTH_OF_FIELD
/**
 * Given a sampled point at [-1,1]x[-1,1], uniformly map to some values on disk
 *  in concentric style
 *
 * @return  (x,y) point on a unit disk
 */
__device__ glm::vec2 concentricSampleDisk(const glm::vec2 &u) {
  // Map uniform random numbers to [-1, 1]x[-1, 1]
  glm::vec2 offset = 2.f * u - glm::vec2(1, 1);

  // Handle degeneracy at the origin
  if (offset.x == 0 && offset.y == 0) return glm::vec2(0, 0);

  // Apply concentric mapping to point
  float theta, r;
  if (std::abs(offset.x) > std::abs(offset.y)) {
    r     = offset.x;
    theta = PI_4 * (offset.y / offset.x);
  } else {
    r     = offset.y;
    theta = PI_2 - PI_4 * (offset.x / offset.y);
  }
  return r * glm::vec2(std::cos(theta), std::sin(theta));
}

/**
 * Updates the origin & direction of each generated ray based on thin-lens
 * camera model Reference:
 * https://www.pbr-book.org/3ed-2018/Camera_Models/Projective_Camera_Models
 *
 * @return  Ray&  ray
 */
__device__ void updateRayOnLens(Camera cam, int iter, int ray_index, int depth,
                                Ray &ray) {
  glm::mat4 view_mat         = glm::lookAt(cam.position, cam.lookAt, cam.up);
  glm::mat4 view_mat_inverse = glm::inverse(view_mat);

  // 1. sample point on len disk
  thrust::default_random_engine rng =
      makeSeededRandomEngine(iter, ray_index, depth);
  thrust::uniform_real_distribution<float> u01(0, 1);
  glm::vec3 pt_on_lens =
      cam.lensRadius *
      glm::vec3(concentricSampleDisk(glm::vec2(u01(rng), u01(rng))), 0.0f);

  // 2. compute intersection of pinhole ray with plane of focus (in camera local
  // coordinates)
  glm::vec3 origin_local = glm::vec3(view_mat * glm::vec4(ray.origin, 1.0f));
  glm::vec3 dir_local    = glm::vec3(view_mat * glm::vec4(ray.direction, 0.0f));
  float ft               = glm::abs(cam.focalDistance / dir_local.z);
  glm::vec3 pt_on_focus_local = origin_local + ft * dir_local;
  glm::vec3 pt_on_focus =
      glm::vec3(view_mat_inverse * glm::vec4(pt_on_focus_local, 1.0f));

  // 3. update ray's origin & direction on point
  glm::vec3 origin_new =
      glm::vec3(view_mat_inverse * glm::vec4(pt_on_lens, 1.0f));
  glm::vec3 dir_new = glm::normalize(pt_on_focus - origin_new);

  // 4. return
  ray.origin    = origin_new;
  ray.direction = dir_new;
}
#endif

/**
 * Generate PathSegments with rays from the camera through the screen into the
 * scene, which is the first bounce of rays.
 *
 * Antialiasing - add rays for sub-pixel sampling
 * motion blur - jitter rays "in time"
 * lens effect - jitter ray origin positions based on a lens
 */
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth,
                                      PathSegment *pathSegments) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < cam.resolution.x && y < cam.resolution.y) {
    int index = x + (y * cam.resolution.x);

    // primary ray per pixel
    PathSegment &segment     = pathSegments[index];
    segment.ray.origin       = cam.position;
    segment.color            = glm::vec3(1.0f, 1.0f, 1.0f);
    segment.pixelIndex       = index;
    segment.remainingBounces = traceDepth;
    segment.ray.direction =
        glm::normalize(cam.view -
                       cam.right * cam.pixelLength.x *
                           ((float)x - (float)cam.resolution.x * 0.5f) -
                       cam.up * cam.pixelLength.y *
                           ((float)y - (float)cam.resolution.y * 0.5f));
#ifdef DEPTH_OF_FIELD
    if (cam.lensRadius > 0) {
      updateRayOnLens(cam, iter, index, INT_MAX, segment.ray);
    }
#endif

    // implement antialiasing by jittering the ray
    // sub-sampled extra rays per pixel
    int pixelcount = cam.resolution.x * cam.resolution.y;
    for (int i = 1; i < ANTIALIAS_FACTOR; ++i) {
      PathSegment &extra_segment     = pathSegments[i * pixelcount + index];
      extra_segment.ray.origin       = cam.position;
      extra_segment.color            = glm::vec3(1.0f, 1.0f, 1.0f);
      extra_segment.pixelIndex       = index;
      extra_segment.remainingBounces = traceDepth;
      thrust::default_random_engine rng =
          makeSeededRandomEngine(iter, index, i);
      thrust::uniform_real_distribution<float> u01(0, 1);
      extra_segment.ray.direction = glm::normalize(
          cam.view -
          cam.right * cam.pixelLength.x *
              ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f) -
          cam.up * cam.pixelLength.y *
              ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f));
#ifdef DEPTH_OF_FIELD
      if (cam.lensRadius > 0) {
        updateRayOnLens(cam, iter, index, i, extra_segment.ray);
      }
#endif
    }
  }
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(const int depth, const int num_paths,
                                     const PathSegment *pathSegments,
                                     const Geom *geoms, const int geoms_size,
                                     ShadeableIntersection *intersections,
                                     int *materialIDs) {
  int path_index = blockIdx.x * blockDim.x + threadIdx.x;

  if (path_index < num_paths) {
    PathSegment pathSegment = pathSegments[path_index];

    float t;
    glm::vec3 intersect_point;
    glm::vec3 normal;
    float t_min        = FLT_MAX;
    int hit_geom_index = -1;
    bool outside       = true;

    glm::vec3 tmp_intersect;
    glm::vec3 tmp_normal;

    // naive parse through global geoms

    for (int i = 0; i < geoms_size; i++) {
      const Geom &geom = geoms[i];

      if (geom.type == CUBE) {
        t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect,
                                tmp_normal, outside);
      } else if (geom.type == SPHERE) {
        t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect,
                                   tmp_normal, outside);
      } else if (geom.type == TRIANGLE) {
        t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect,
                                     tmp_normal, outside);
      }

      // Compute the minimum t from the intersection tests to determine what
      // scene geometry object was hit first.
      if (t > 0.0f && t_min > t) {
        t_min           = t;
        hit_geom_index  = i;
        intersect_point = tmp_intersect;
        normal          = tmp_normal;
      }
    }

    if (hit_geom_index == -1) {
      intersections[path_index].t = -1.0f;
      materialIDs[path_index]     = -1;
    } else {
      // The ray hits something
      int material_id                      = geoms[hit_geom_index].materialid;
      intersections[path_index].t          = t_min;
      intersections[path_index].materialId = material_id;
      intersections[path_index].surfaceNormal = normal;
      materialIDs[path_index]                 = material_id;
    }
  }
}

/**
 * Shade all the intersections according to materials using BSDF.
 *
 * @return  updates pathSegments
 */
__global__ void shadeMaterial(
    int iter, int depth, int num_paths,
    const ShadeableIntersection *shadeableIntersections,
    const Material *materials, PathSegment *pathSegments) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths) {
    const ShadeableIntersection intersection = shadeableIntersections[idx];
    PathSegment path_segment                 = pathSegments[idx];
    if (intersection.t > 0.0f) {  // if the intersection exists
      // Set up an random number generator
      thrust::default_random_engine rng =
          makeSeededRandomEngine(iter, idx, depth);

      const Material material = materials[intersection.materialId];
      glm::vec3 intersect_pos = getPointOnRay(path_segment.ray, intersection.t);
      scatterRay(path_segment, intersect_pos, intersection.surfaceNormal,
                 material, rng);
    }
    // If there was no intersection, color the ray black, terminates bouncing
    else {
      path_segment.color            = glm::vec3(0.0f);
      path_segment.remainingBounces = 0;
    }
    pathSegments[idx] = path_segment;
  }
}

__global__ void generateGBuffer(int num_paths,
                                ShadeableIntersection *shadeableIntersections,
                                PathSegment *pathSegments,
                                GBufferPixel *gBuffer) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths) {
    gBuffer[idx].position =
        getPointOnRay(pathSegments[idx].ray, shadeableIntersections[idx].t);
    gBuffer[idx].t      = shadeableIntersections[idx].t;
    gBuffer[idx].normal = shadeableIntersections[idx].surfaceNormal;
  }
}

// Add the current iteration's output to the image buffer
__global__ void finalGather(int nPaths, glm::vec3 *img_buffer,
                            const PathSegment *iterationPaths) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < nPaths) {
    PathSegment iterationPath = iterationPaths[index];
    atomicAdd(&img_buffer[iterationPath.pixelIndex][0], iterationPath.color[0]);
    atomicAdd(&img_buffer[iterationPath.pixelIndex][1], iterationPath.color[1]);
    atomicAdd(&img_buffer[iterationPath.pixelIndex][2], iterationPath.color[2]);
  }
}

// Average the accumulative subpixel values in image buffer & add it to final
// image
__global__ void addToImage(int pixelcount, glm::vec3 *image,
                           const glm::vec3 *img_buffer) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < pixelcount) {
    image[index] += (img_buffer[index] / (1.0f * ANTIALIAS_FACTOR));
  }
}

__global__ void atrousDenoiser(glm::vec3 *image_denoised, float *weights,
                               float *posWeights, float *colorWeights,
                               float *norWeights, const glm::vec3 *image,
                               const glm::ivec2 resolution, const float c_phi,
                               const float n_phi, const float p_phi,
                               const int stepwidth,
                               const glm::ivec2 *kernel_offset,
                               const GBufferPixel *gBuffer) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x < resolution.x && y < resolution.y) {
    int pix_index = x + (y * resolution.x);
    glm::vec3 sum{0.f, 0.f, 0.f};
    float sum_weight = 0.f;

    // for debug visualizations
    float sum_w_normal = 0.f;
    float sum_w_pos    = 0.f;
    float sum_w_color  = 0.f;

    GBufferPixel pix_gBuffer = gBuffer[pix_index];
    glm::vec3 pix_color      = image[pix_index];
    glm::vec3 pix_normal     = pix_gBuffer.normal;
    glm::vec3 pix_pos        = pix_gBuffer.position;

    for (int i = 0; i < KERNEL_SIZE; ++i) {
      glm::ivec2 adj_xy =
          glm::clamp(glm::ivec2(x, y) + stepwidth * kernel_offset[i],
                     glm::ivec2(0, 0), resolution - glm::ivec2(1, 1));
      int adj_index   = adj_xy.x + (adj_xy.y * resolution.x);
      GBufferPixel gb = gBuffer[adj_index];

      glm::vec3 color  = image[adj_index];
      float color_dist = glm::length(pix_color - color);
      float w_color = glm::min(glm::exp(-(color_dist) / (c_phi * c_phi)), 1.0f);

      glm::vec3 normal  = gb.normal;
      float normal_dist = glm::length(pix_normal - normal);
      float w_normal =
          glm::min(glm::exp(-(normal_dist) / (n_phi * n_phi)), 1.0f);

      glm::vec3 pos  = gb.position;
      float pos_dist = glm::length(pix_pos - pos);
      float w_pos    = glm::min(glm::exp(-(pos_dist) / (p_phi * p_phi)), 1.0f);

      float weight = w_color * w_normal * w_pos;
      sum += color * weight * cdev_kernel[i];

      // for debug visualizations
      sum_weight += weight * cdev_kernel[i];
      sum_w_pos += w_pos;
      sum_w_normal += w_normal;
      sum_w_color += w_color;
    }
    image_denoised[pix_index] = sum / sum_weight;

    // for debug visualizations
    weights[pix_index]      = sum_weight;
    posWeights[pix_index]   = sum_w_pos / KERNEL_SIZE;
    norWeights[pix_index]   = sum_w_normal / KERNEL_SIZE;
    colorWeights[pix_index] = sum_w_color / KERNEL_SIZE;
  }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(int frame, int iteration) {
  const int traceDepth = hst_scene->state.traceDepth;
  const Camera &cam    = hst_scene->state.camera;
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // 2D block for generating ray from camera
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // 1D block for path tracing
  const int blockSize1d = 128;

  ///////////////////////////////////////////////////////////////////////////
  // Recap:
  // * Initialize array of path rays (using rays that come out of the camera)
  //   * You can pass the Camera object to that kernel.
  //   * Each path ray must carry at minimum a (ray, color) pair,
  //   * where color starts as the multiplicative identity, white = (1, 1, 1).
  //   * This has already been done for you.
  // * For each depth:
  //   * Compute an intersection in the scene for each path ray.
  //     A very naive version of this has been implemented for you, but feel
  //     free to add more primitives and/or a better algorithm.
  //     Currently, intersection distance is recorded as a parametric distance,
  //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
  //     * Color is attenuated (multiplied) by reflections off of any object
  //   * Stream compact away all of the terminated paths.
  //     You may use either your implementation or `thrust::remove_if` or its
  //     cousins.
  //     * Note that you can't really use a 2D kernel launch any more - switch
  //       to 1D.
  //   * Shade the rays that intersected something or didn't bottom out.
  //     That is, color the ray by performing a color computation according
  //     to the shader, then generate a new ray to continue the ray path.
  //     We recommend just updating the ray's PathSegment in place.
  //     Note that this step may come before or after stream compaction,
  //     since some shaders you write may also cause a path to terminate.
  // * Finally:
  //     * if not denoising, add this iteration's results to the image
  //     * if denoising, run kernels that take both the raw pathtraced
  //        result and the gbuffer, and put the result in the "pbo" from opengl
  ///////////////////////////////////////////////////////////////////////////

  generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(
      cam, iteration, traceDepth, dev_paths);
  checkCUDAError("generate camera ray");

  int depth            = 0;
  int num_active_paths = ANTIALIAS_FACTOR * pixelcount;

  // Empty gbuffer
  hipMemset(dev_gBuffer, 0, pixelcount * sizeof(GBufferPixel));

  // --- PathSegment Tracing Stage ---
  // Shoot ray into scene, bounce between objects, push shading chunks
  while (num_active_paths > 0) {
    // clean shading chunks
    hipMemset(dev_intersections, 0,
               ANTIALIAS_FACTOR * pixelcount * sizeof(ShadeableIntersection));

    // --- Tracing Stage ---
    dim3 numblocksPathSegmentTracing =
        (num_active_paths + blockSize1d - 1) / blockSize1d;

#ifdef CACHE_INTERSECTIONS
    if (depth == 0 && iteration > 1) {
      hipMemcpy(dev_intersections, dev_intersections_cache,
                 pixelcount * sizeof(ShadeableIntersection),
                 hipMemcpyDeviceToDevice);
      hipMemcpy(dev_materialIDs, dev_materialIDs_cache,
                 pixelcount * sizeof(int), hipMemcpyDeviceToDevice);
      if (num_active_paths - pixelcount > 0) {
        dim3 numBlocksAntialiasTracing =
            (num_active_paths - pixelcount + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numBlocksAntialiasTracing, blockSize1d>>>(
            depth, num_active_paths - pixelcount, dev_paths + pixelcount,
            dev_geoms, hst_scene->geoms.size(), dev_intersections + pixelcount,
            dev_materialIDs + pixelcount);
        checkCUDAError("anti-alias extra rays trace one bounce");
        hipDeviceSynchronize();
      }
    } else {
      computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
          depth, num_active_paths, dev_paths, dev_geoms,
          hst_scene->geoms.size(), dev_intersections, dev_materialIDs);
      checkCUDAError("trace one bounce");
      hipDeviceSynchronize();

      if (depth == 0 && iteration == 1) {
        hipMemcpy(dev_intersections_cache, dev_intersections,
                   pixelcount * sizeof(ShadeableIntersection),
                   hipMemcpyDeviceToDevice);
        hipMemcpy(dev_materialIDs_cache, dev_materialIDs,
                   pixelcount * sizeof(int), hipMemcpyDeviceToDevice);
      }
    }
#else
    computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
        depth, num_active_paths, dev_paths, dev_geoms, hst_scene->geoms.size(),
        dev_intersections, dev_materialIDs);
    checkCUDAError("trace one bounce");
    hipDeviceSynchronize();
#endif

    if (depth == 0) {
      dim3 numBlocksGBuffer = (pixelcount + blockSize1d - 1) / blockSize1d;
      generateGBuffer<<<numBlocksGBuffer, blockSize1d>>>(
          pixelcount, dev_intersections, dev_paths, dev_gBuffer);
    }

    depth++;

    // --- Shading Stage ---
    // Shade path segments based on intersections and generate new rays by
    // evaluating the BSDF.
    // Start off with just a big kernel that handles all the different
    // materials you have in the scenefile.
    // Compare between directly shading the path segments and shading
    // path segments that have been reshuffled to be contiguous in memory.
    //
    // a) sort intersections & paths with ray material ID
    if (static_config::enable_radixSort) {
      hipMemcpy(dev_materialIDBuffers, dev_materialIDs,
                 num_active_paths * sizeof(int), hipMemcpyDeviceToDevice);
      thrust::sort_by_key(thrust::device, dev_materialIDs,
                          dev_materialIDs + num_active_paths,
                          dev_intersections);
      thrust::sort_by_key(thrust::device, dev_materialIDBuffers,
                          dev_materialIDBuffers + num_active_paths, dev_paths);
    }

    // b) Use BSDF to shade & advance each path segment
    shadeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
        iteration, depth, num_active_paths, dev_intersections, dev_materials,
        dev_paths);
    checkCUDAError("shade material");
    hipDeviceSynchronize();

    // c) Stream-compact & discard the terminated path segments
    num_active_paths =
        stream_compaction::rayCompaction(dev_paths, num_active_paths);
    checkCUDAError("ray compaction");
  }

  // Assemble this iteration and apply it to the image
  hipMemset(dev_image_buffer, 0, pixelcount * sizeof(glm::vec3));
  dim3 numBlocksSubPixels =
      (ANTIALIAS_FACTOR * pixelcount + blockSize1d - 1) / blockSize1d;
  finalGather<<<numBlocksSubPixels, blockSize1d>>>(
      ANTIALIAS_FACTOR * pixelcount, dev_image_buffer, dev_paths);
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
  addToImage<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image,
                                               dev_image_buffer);

  checkCUDAError("pathtrace");
}

// CHECKITOUT: this kernel "post-processes" the gbuffer/gbuffers into something
// that you can visualize for debugging.
void showGBufferNormal(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  // CHECKITOUT: process the gbuffer results and send them to OpenGL buffer for
  // visualization
  gbufferNormalToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution,
                                                       dev_gBuffer);
}

void showGBufferPosition(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  gbufferPositionToPBO<<<blocksPerGrid2d, blockSize2d>>>(
      pbo, cam.resolution, dev_gBuffer, hst_scene->boundary.min_xyz,
      hst_scene->boundary.max_xyz);
}

void showGBufferWeights(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  gbufferWeightToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution,
                                                       dev_weights);
}

void showGBufferPositionWeights(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  gbufferWeightToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution,
                                                       dev_posWeights);
}

void showGBufferNormalWeights(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  gbufferWeightToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution,
                                                       dev_norWeights);
}

void showGBufferColorWeights(uchar4 *pbo) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

  gbufferWeightToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution,
                                                       dev_colorWeights);
}

void showImage(uchar4 *pbo, int iter) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // Send results to OpenGL buffer for rendering
  sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter,
                                                   dev_image);
  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image,
             pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy from dev_image to scene");
}

void denoiseImage(int filter_width, float c_phi, float n_phi, float p_phi) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  hipMemcpy(dev_image_denoised, dev_image, pixelcount * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);

  int stepwidth    = 1;
  int kernel_width = KERNEL_WIDTH;
  while (kernel_width < filter_width) {
    atrousDenoiser<<<blocksPerGrid2d, blockSize2d>>>(
        dev_image_denoised_buffer, dev_weights, dev_posWeights,
        dev_colorWeights, dev_norWeights, dev_image_denoised, cam.resolution,
        c_phi, n_phi, p_phi, stepwidth, dev_kernelOffset, dev_gBuffer);
    std::swap(dev_image_denoised, dev_image_denoised_buffer);
    stepwidth++;
    kernel_width = (KERNEL_WIDTH - 1) * stepwidth + 1;
    c_phi /= 2.0f;
  }
}

void showDenoisedImage(uchar4 *pbo, int iter) {
  const Camera &cam = hst_scene->state.camera;
  const dim3 blockSize2d(8, 8);
  const dim3 blocksPerGrid2d(
      (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
      (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);
  const int pixelcount = cam.resolution.x * cam.resolution.y;

  // Send results to OpenGL buffer for rendering
  sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter,
                                                   dev_image_denoised);
  // Retrieve image from GPU
  hipMemcpy(hst_scene->state.image.data(), dev_image_denoised,
             pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
  checkCUDAError("hipMemcpy from dev_image_denoised to scene");
}
