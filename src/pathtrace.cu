#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/device_vector.h>
#include <thrust/partition.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define ERRORCHECK 1
#define ANTI_ALIASING 0
#define CACHE_BOUNCE 0
#define SORT_MATERIALS 0
#define DEPTH_OF_FIELD 0
#define DIRECT_LIGHTING 1
#define TIME_RENDER 1

#define LENS_RADIUS 0.07
#define FOCAL_DISTANCE 5

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
        int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int) (pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int) (pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int) (pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene * hst_scene = NULL;
static glm::vec3 * dev_image = NULL;
static Geom * dev_geoms = NULL;
static Material * dev_materials = NULL;
static PathSegment * dev_paths = NULL;
static ShadeableIntersection * dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
// ...
static ShadeableIntersection* dev_first_bounce = NULL;
#if DIRECT_LIGHTING
static Geom* dev_lights = NULL;
#endif

void pathtraceInit(Scene *scene) {
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
#if CACHE_BOUNCE || SORT_MATERIALS
    hipMalloc(&dev_first_bounce, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_first_bounce, 0, pixelcount * sizeof(ShadeableIntersection));
#endif

#if DIRECT_LIGHTING
    hipMalloc(&dev_lights, scene->lights.size() * sizeof(Geom));
    hipMemcpy(dev_lights, scene->lights.data(), scene->lights.size() * sizeof(Geom), hipMemcpyHostToDevice);
#endif

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
#if CACHE_BOUNCE || SORT_MATERIALS
    hipFree(dev_first_bounce);
#endif

#if DIRECT_LIGHTING
    hipFree(dev_lights);
#endif
    checkCUDAError("pathtraceFree");
}

__host__ __device__
glm::vec3 pointOnPlane(Geom light, thrust::default_random_engine& rng) {
    thrust::uniform_real_distribution<float> u01(0, 1);
    glm::vec2 pt(u01(rng), u01(rng));
    glm::vec3 planePt = glm::vec3((pt - glm::vec2(0.5f)), 0.f);
    return glm::vec3(light.transform * glm::vec4(planePt, 1.f));
}


__host__ __device__
glm::vec3 convertDisk(const glm::vec2 &v) {
    float x = v.x;
    float y = v.y;
    float phi, r;
    float a = 2 * x - 1.f;
    float b = 2 * y - 1.f;

    if (a > -b) {
        if (a > b) {
            r = a;
            phi = (PI / 4) * (b / a);
        }
        else {
            r = b;
            phi = (PI / 4) * (2 - (a / b));
        }
    }
    else {
        if (a < b) {
            r = -a;
            phi = (PI / 4) * (4 + (b / a));
        }
        else {
            r = -b;
            if (b < 0 || b > 0) {
                phi = (PI / 4) * (6 - (a / b));
            }
            else {
                phi = 0;
            }
        }
    }
    return glm::vec3(cosf(phi) * r, sinf(phi) * r, 0);
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment & segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        //antialiasing by jittering the ray
        float x_aa = x;
        float y_aa = y;
        thrust::default_random_engine random = makeSeededRandomEngine(iter, index, traceDepth);
        

#if ANTI_ALIASING
        thrust::uniform_real_distribution<float> u01(-0.55f, 0.55f);
        x_aa += u01(random);
        y_aa += u01(random);
#endif
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x_aa - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y_aa - (float)cam.resolution.y * 0.5f)
            );

        //for depth of field
#if DEPTH_OF_FIELD
        thrust::uniform_real_distribution<float> u02(0, 1);
        glm::vec3 sample = convertDisk(glm::vec2(u02(random), u02(random)));
        glm::vec3 lens = (float)LENS_RADIUS * sample;
        glm::vec3 pt = segment.ray.origin + lens;
        glm::vec3 fp = segment.ray.origin + (float)FOCAL_DISTANCE * segment.ray.direction;

        segment.ray.origin = pt;
        segment.ray.direction = glm::normalize(fp - pt);

#endif
        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth
    , int num_paths
    , PathSegment * pathSegments
    , Geom * geoms
    , int geoms_size
    , ShadeableIntersection * intersections
    )
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom & geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == TRIANGLE) {
                t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// shade for direct lighting
__global__ void shadeDirectLighting(
    int iter
    , int num_paths
    , ShadeableIntersection* shadeableIntersections
    , PathSegment* pathSegments
    , Material* materials
    , Geom* lights
    , int num
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        if (pathSegments[idx].remainingBounces <= 0) {
            return;
        }

        ShadeableIntersection intersection = shadeableIntersections[idx];
        thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
        PathSegment path = pathSegments[idx];

        if (path.remainingBounces != 2 && path.remainingBounces > 0 && intersection.t > 0.f) {

            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            else if (pathSegments[idx].remainingBounces == 1) {
                pathSegments[idx].remainingBounces -= 1;
                pathSegments[idx].color = glm::vec3(0.0f);
            }
            else {
                pathSegments[idx].remainingBounces -= 1;
                scatterRay(pathSegments[idx], pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t, intersection.surfaceNormal,
                    material, rng);
            }
            
        }
        else if (path.remainingBounces == 2 && intersection.t > 0.f) {
            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;
            }
            else {
                scatterRay(path, path.ray.origin + path.ray.direction * intersection.t, intersection.surfaceNormal, material, rng);
                thrust::uniform_real_distribution<float> u01(0, 1);
                float r = u01(rng);
                int lightIdx = 0;
                if (num != 0) {
                    lightIdx = glm::min((int)glm::floor(r * num), num - 1);
                }
                glm::vec3 lightPt = pointOnPlane(lights[lightIdx], rng);
                path.ray.direction = glm::normalize(lightPt - path.ray.origin);
                path.remainingBounces--;
            }
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
            pathSegments[idx].remainingBounces = 0;
        }
    }
}


__global__ void shadeFakeMaterial (
  int iter
  , int num_paths
    , ShadeableIntersection * shadeableIntersections
    , PathSegment * pathSegments
    , Material * materials
    )
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_paths)
  {
      if (pathSegments[idx].remainingBounces <= 0) {
          return;
      }

    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f) { // if the intersection exists...
      // Set up the RNG
      // LOOK: this is how you use thrust's RNG! Please look at
      // makeSeededRandomEngine as well.
      thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
      thrust::uniform_real_distribution<float> u01(0, 1);

      Material material = materials[intersection.materialId];
      glm::vec3 materialColor = material.color;

      // If the material indicates that the object was a light, "light" the ray
      if (material.emittance > 0.0f) {
        pathSegments[idx].color *= (materialColor * material.emittance);
        pathSegments[idx].remainingBounces = 0;
      }
      else if (pathSegments[idx].remainingBounces == 1) {
          pathSegments[idx].remainingBounces -= 1;
          pathSegments[idx].color = glm::vec3(0.0f);
      }
      else {
          pathSegments[idx].remainingBounces -= 1;
          scatterRay(pathSegments[idx], pathSegments[idx].ray.origin + pathSegments[idx].ray.direction * intersection.t, intersection.surfaceNormal,
              material, rng);
      }
    // If there was no intersection, color the ray black.
    } else {
      pathSegments[idx].color = glm::vec3(0.0f);
      pathSegments[idx].remainingBounces = 0;
    }
  }
}


// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 * image, PathSegment * iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

struct should_end {
    __host__ __device__
        bool operator()(const PathSegment& pathSegment) {
        return (pathSegment.remainingBounces >= 0);
    }
};

struct compare_materials {
    __host__ __device__
        bool operator()(const ShadeableIntersection& m1, const ShadeableIntersection& m2) {
        return (m1.materialId > m2.materialId);
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
            (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
            (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera <<<blocksPerGrid2d, blockSize2d >>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

  bool iterationComplete = false;
    while (!iterationComplete) {

    // tracing
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
    //cache first bounce
    if (CACHE_BOUNCE && !ANTI_ALIASING && depth == 0 && iter != 1) {
        thrust::copy(thrust::device, dev_first_bounce, dev_first_bounce + num_paths, dev_intersections);

        //sort by material
        if (SORT_MATERIALS) {
            thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compare_materials());
        }
    }
    else {

        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections);
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();

        //cache first bounce
        if (CACHE_BOUNCE && !ANTI_ALIASING && depth == 0 && iter == 1) {
            thrust::copy(thrust::device, dev_intersections, dev_intersections + num_paths, dev_first_bounce);
        }
        //sort by material
        if (SORT_MATERIALS) {
            thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, compare_materials());
        }
    }

    depth++;

#if DIRECT_LIGHTING
    shadeDirectLighting<<<numblocksPathSegmentTracing, blockSize1d>>>(iter, num_paths, dev_intersections, dev_paths, 
                                                                            dev_materials, dev_lights, hst_scene->lights.size());
#else
  shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>> (iter, num_paths, dev_intersections, dev_paths, dev_materials);
#endif

  dev_path_end = thrust::stable_partition(thrust::device, dev_paths, dev_path_end, should_end());
  num_paths = dev_path_end - dev_paths;


    if (num_paths == 0 || depth > traceDepth) {
        iterationComplete = true;
    }
        
}

  // Assemble this iteration and apply it to the image
  dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
            pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
