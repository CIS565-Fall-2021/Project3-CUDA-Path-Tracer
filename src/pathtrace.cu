#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include "hip/hip_math_constants.h"

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#define SORT_MATERIAL 1
#define CACHE_INTERSECTION 0
#define DEPTH_OF_FIELD 1
#define MESH_BOUND_CHECK 1
#define ANTI_ALIASING 0

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line) {
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err) {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file) {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#  ifdef _WIN32
    getchar();
#  endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution,
    int iter, glm::vec3* image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y) {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static ShadeableIntersection* dev_cache_intersections = NULL;
//Triangles for testing arbitrary obj file intersection
static TriangleGeom* dev_triangles = NULL;

void pathtraceInit(Scene* scene) {
    hst_scene = scene;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_cache_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_triangles, scene->triangles.size() * sizeof(TriangleGeom));
    hipMemcpy(dev_triangles, scene->triangles.data(), scene->triangles.size() * sizeof(TriangleGeom), hipMemcpyHostToDevice);

    checkCUDAError("pathtraceInit");
}

void pathtraceFree() {
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_cache_intersections);
    hipFree(dev_triangles);
    checkCUDAError("pathtraceFree");
}



__host__ __device__ glm::vec2 concentricSampleDisk(thrust::default_random_engine& rng) {

    thrust::uniform_real_distribution<float> u01(0, 1);

    glm::vec2 u = glm::vec2(u01(rng), u01(rng));
    glm::vec2 uOffset = 2.f * u - glm::vec2(1.f, 1.f);

    if (uOffset.x == 0 && uOffset.y == 0) {
        return glm::vec2(0.f, 0.f);
    }
    float theta, r;
    if (std::abs(uOffset.x) > std::abs(uOffset.y)) {
        r = uOffset.x;
        theta = HIP_PIO4_F * (uOffset.y / uOffset.x);
    }
    else {
        r = uOffset.y;
        theta = HIP_PIO2_F - HIP_PIO4 * (uOffset.x / uOffset.y);
    }
    return r * glm::vec2(std::cos(theta), std::sin(theta));
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, segment.remainingBounces);

#if ANTI_ALIASING
        thrust::uniform_real_distribution<float> u01(-0.5, 0.5);
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + u01(rng) - (float)cam.resolution.y * 0.5f)
        );
#else
        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y - (float)cam.resolution.y * 0.5f)
        );
#endif

#if DEPTH_OF_FIELD
        cam.focalDistance = 5.f;
        cam.lensRadius = 0.2f;

        if (cam.lensRadius > 0) {
            glm::vec2 randomSample{ 0 };
            glm::vec2 pLens = cam.lensRadius * concentricSampleDisk(rng);

            float ft = glm::abs((cam.focalDistance) / segment.ray.direction.z);
            glm::vec3 pFocus = ft * segment.ray.direction;

            segment.ray.origin += glm::vec3(pLens.x, pLens.y, 0.f);
            segment.ray.direction = glm::normalize(pFocus - glm::vec3(pLens.x, pLens.y, 0.f));
        }
#endif

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

__host__ __device__ bool checkWithinMeshBoundingBox(Geom& object, 
    glm::vec3 min, glm::vec3 max, Ray& r) {
    //Basically the same thing as boxIntersectionTest
    Ray q;
    q.origin = multiplyMV(object.inverseTransform, glm::vec4(r.origin, 1.0f));
    q.direction = glm::normalize(multiplyMV(object.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz) {
        float qdxyz = q.direction[xyz];
        if (glm::abs(qdxyz) > 0.00001f) {
            float t1 = (min[xyz] - q.origin[xyz]) / qdxyz;
            float t2 = (max[xyz] - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin) {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax) {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0) {
        return true;
    }
    return false;
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth
    , int num_paths
    , PathSegment* pathSegments
    , Geom* geoms
    , int geoms_size
    , TriangleGeom* triangles
    , int triangles_size
    , glm::vec3 triangle_bound_min 
    , glm::vec3 triangle_bound_max
    , ShadeableIntersection* intersections
)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == CUSTOM_OBJ) {
                //Compute the minimum t (a ray might intersect multiple triangle (front and back) in a custom obj
#if MESH_BOUND_CHECK
                //Only spawn the rays which hits inside the bounding box
                if (checkWithinMeshBoundingBox(geom, triangle_bound_min, triangle_bound_max, pathSegment.ray)) {
                    float closest_dist = FLT_MAX;
                    for (int j = 0; j < triangles_size; j++) {
                        TriangleGeom& triangle = triangles[j];
                        float triangle_inter = triangleIntersectionTest(geom, pathSegment.ray,
                            tmp_intersect, triangle.vertex1, triangle.vertex2, triangle.vertex3,
                            triangle.normal1, triangle.normal2, triangle.normal3, tmp_normal, outside);
                        if (triangle_inter != -1) {
                            closest_dist = glm::min(closest_dist, triangle_inter);
                        }
                    }
                    t = closest_dist;
                }
#else
                float closest_dist = FLT_MAX;
                for (int j = 0; j < triangles_size; j++) {
                    TriangleGeom& triangle = triangles[j];
                    float triangle_inter = triangleIntersectionTest(geom, pathSegment.ray,
                        tmp_intersect, triangle.vertex1, triangle.vertex2, triangle.vertex3,
                        triangle.normal1, triangle.normal2, triangle.normal3, tmp_normal, outside);
                    if (triangle_inter != -1) {
                        closest_dist = glm::min(closest_dist, triangle_inter);
                    }
                }
                t = closest_dist;
#endif
            }
            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                normal = tmp_normal;
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter
    , int num_paths
    , ShadeableIntersection* shadeableIntersections
    , PathSegment* pathSegments
    , Material* materials) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) { // if the intersection exists...
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, 
                pathSegments[idx].remainingBounces);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                pathSegments[idx].remainingBounces = 0;
                pathSegments[idx].color *= (materialColor * material.emittance);
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            // TODO: replace this! you should be able to start with basically a one-liner
            else {
                scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t), 
                    intersection.surfaceNormal, material, rng);
                //float lightTerm = glm::dot(intersection.surfaceNormal, glm::vec3(0.0f, 1.0f, 0.0f));
                //pathSegments[idx].color *= (materialColor * lightTerm) * 0.3f + ((1.0f - intersection.t * 0.02f) * materialColor) * 0.7f;
                //pathSegments[idx].color *= u01(rng); // apply some noise because why not
            }
            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].remainingBounces = 0;
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

struct needCompact
{
    __host__ __device__ bool operator()(const PathSegment& seg)
    {
        return seg.remainingBounces;
    }
};

struct cmpMaterials
{
    //overwrite sort_by_key compares key objects using operator<.
    __host__ __device__ bool operator()(ShadeableIntersection& intersect1, const ShadeableIntersection& intersect2)
    {
        return intersect1.materialId < intersect2.materialId;
    }
};

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter) {
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera << <blocksPerGrid2d, blockSize2d >> > (cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete) {
    dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
       
#if CACHE_INTERSECTION
        
    if (iter == 1 || depth != 0) {
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

        glm::vec3 check = hst_scene->triangle_bound_min;
        glm::vec3 check2 = hst_scene->triangle_bound_max;

        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth
            , num_paths
            , dev_paths
            , dev_geoms
            , hst_scene->geoms.size()
            , dev_triangles
            , hst_scene->triangles.size()
            , hst_scene->triangle_bound_min
            , hst_scene->triangle_bound_max
            , dev_intersections
            );
        checkCUDAError("trace one bounce");

        hipDeviceSynchronize();
        if (depth == 0) {
            // store first time bounce into cache
            hipMemcpy(dev_cache_intersections, dev_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyHostToHost);
            hipDeviceSynchronize();
        }
    }
    else {
        hipMemcpy(dev_intersections, dev_cache_intersections, pixelcount * sizeof(ShadeableIntersection), hipMemcpyHostToHost);
    }
        
#else 
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        computeIntersections << <numblocksPathSegmentTracing, blockSize1d >> > (
            depth
            , num_paths
            , dev_paths
            , dev_geoms
            , hst_scene->geoms.size()
            , dev_triangles
            , hst_scene->triangles.size()
            , hst_scene->triangle_bound_min
            , hst_scene->triangle_bound_max
            , dev_intersections
            );
        checkCUDAError("trace one bounce");

        hipDeviceSynchronize();
#endif
        depth++;

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
      // evaluating the BSDF.
      // Start off with just a big kernel that handles all the different
      // materials you have in the scenefile.
      // TODO: compare between directly shading the path segments and shading
      // path segments that have been reshuffled to be contiguous in memory.

#if SORT_MATERIAL
        thrust::sort_by_key(thrust::device, dev_intersections, dev_intersections + num_paths, dev_paths, cmpMaterials());
#endif

        shadeFakeMaterial << <numblocksPathSegmentTracing, blockSize1d >> > (
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials
            );

        // compaction using thrust::partition
        dev_path_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, needCompact());
        num_paths = dev_path_end - dev_paths;

        if (num_paths == 0) {
            iterationComplete = true; // TODO: should be based off stream compaction results.
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather << <numBlocksPixels, blockSize1d >> > (pixelcount, dev_image, dev_paths);

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO << <blocksPerGrid2d, blockSize2d >> > (pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");

}
