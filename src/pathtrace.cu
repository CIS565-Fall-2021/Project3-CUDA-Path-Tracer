#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "pathtrace.h"
#include "intersections.h"
#include "interactions.h"

#include <chrono>

#define ERRORCHECK 1

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char *msg, const char *file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif
    exit(EXIT_FAILURE);
#endif
}

__host__ __device__
    thrust::default_random_engine
    makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4 *pbo, glm::ivec2 resolution,
                               int iter, glm::vec3 *image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene *hst_scene = NULL;
static glm::vec3 *dev_image = NULL;
static Geom *dev_geoms = NULL;
static Material *dev_materials = NULL;
static PathSegment *dev_paths = NULL;
static ShadeableIntersection *dev_intersections = NULL;
// TODO: static variables for device memory, any extra info you need, etc
static ShadeableIntersection *dev_intersectionCache = NULL;
static struct TexData *dev_texData = NULL;
static struct Triangle *dev_tris = NULL;
static int *dev_backWidth = NULL;
static int *dev_backHeight = NULL;
static glm::vec3 *dev_background = NULL;
// ...

void pathtraceInit(Scene *scene)
{
    hst_scene = scene;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    // TODO: initialize any extra device memeory you need
    hipMalloc(&dev_intersectionCache, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersectionCache, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_texData, scene->texData.size() * sizeof(struct TexData));
    hipMemcpy(dev_texData, scene->texData.data(), scene->texData.size() * sizeof(struct TexData), hipMemcpyHostToDevice);

    hipMalloc(&dev_tris, scene->triangles.size() * sizeof(struct Triangle));
    hipMemcpy(dev_tris, scene->triangles.data(), scene->triangles.size() * sizeof(struct Triangle), hipMemcpyHostToDevice);

    if (scene->backTex.size() > 0)
    {
        hipMalloc(&dev_background, scene->backTex.size() * sizeof(glm::vec3));
        hipMemcpy(dev_background, scene->backTex.data(), scene->backTex.size() * sizeof(glm::vec3), hipMemcpyHostToDevice);
        hipMalloc(&dev_backHeight, sizeof(int));
        hipMemcpy(dev_backHeight, &(scene->backHeight), sizeof(int), hipMemcpyHostToDevice);
        hipMalloc(&dev_backWidth, sizeof(int));
        hipMemcpy(dev_backWidth, &(scene->backWidth), sizeof(int), hipMemcpyHostToDevice);
    }

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{

    hipFree(dev_image); // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    // TODO: clean up any extra device memory you created
    hipFree(dev_intersectionCache);
    hipFree(dev_texData);
    hipFree(dev_tris);
    if (dev_background != NULL)
    {
        hipFree(dev_background);
        hipFree(dev_backHeight);
        hipFree(dev_backWidth);
    }

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment *pathSegments, int depth)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y)
    {

        int index = x + (y * cam.resolution.x);
        PathSegment &segment = pathSegments[index];

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, depth);
        thrust::uniform_real_distribution<float> u01(0, 1);

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        // TODO: implement antialiasing by jittering the ray
        segment.ray.direction = glm::normalize(
            cam.view -
            cam.right * cam.pixelLength.x * ((float)x
#ifdef ANTIALIASING
                                             + u01(rng) - 0.5f
#endif
                                             - (float)cam.resolution.x * 0.5f) -
            cam.up * cam.pixelLength.y * ((float)y
#ifdef ANTIALIASING
                                          + u01(rng) - 0.5f
#endif
                                          - (float)cam.resolution.y * 0.5f));

#ifdef ANTIALIASING
        segment.ray.direction = glm::normalize(
            cam.view - cam.right * cam.pixelLength.x * ((float)x + u01(rng) - 0.5f - (float)cam.resolution.x * 0.5f) -
            cam.up * cam.pixelLength.y * ((float)y + u01(rng) - 0.5f - (float)cam.resolution.y * 0.5f));
#endif

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// TODO:
// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth, int num_paths, PathSegment *pathSegments, Geom *geoms, int geoms_size, ShadeableIntersection *intersections, struct Triangle *tri, struct TexData *texArr, Material *mats)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        float t_min = FLT_MAX;
        int hit_geom_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec2 tmp_uv;
        glm::vec2 uv;

        glm::mat3 tan2ObjMat;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom &geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                tmp_uv = glm::vec2(-1.f);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
                tmp_uv = glm::vec2(-1.f);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?
            else if (geom.type == MESH)
            {
                t = meshIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv, tri, tan2ObjMat);
            }
            // else if (geom.type == TRIANGLE)
            // {
            //     t = triangleIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside, tmp_uv, dev_tris);
            // }

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                hit_geom_index = i;
                intersect_point = tmp_intersect;
                // uv = geom.type == TRIANGLE ? tmp_uv : glm::vec2(-1.f);
                uv = tmp_uv;
                // normal = tmp_normal;
                // sorry not sorry this got out of control
                normal = geom.type == MESH && geom.useTexture
                             ? glm::vec3(
                                   // transform from obj space to scene space
                                   glm::normalize(
                                       geom.invTranspose *
                                       glm::vec4(
                                           // transform from tan space to obj space
                                           (tan2ObjMat *
                                            // convert normal to glmvec3 from -1 to 1
                                            (2.f * texCol2Color(
                                                       // get normal from normmap
                                                       texArr[uv2Idx(
                                                                  uv,
                                                                  mats[geom.materialid].texWidth,
                                                                  mats[geom.materialid].texHeight)]
                                                           .bump) -
                                             glm::vec3(1.f))),
                                           0.f)))
                             : tmp_normal;
                //                          normalTri *= (outside ? 1.f : -1.f);
                // normal = glm::normalize(multiplyMV(tri.invTranspose, glm::vec4(normalTri, 0.f)));
            }
        }

        if (hit_geom_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            //The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = geoms[hit_geom_index].materialid;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].uvs = uv;
            intersections[path_index].useTexture = geoms[hit_geom_index].useTexture;
        }
    }
}

/**
 * @brief Shader that DOES do a BSDF Evaluation
 * 
 * @param iter iteration number
 * @param num_paths total number of paths/rays
 * @param shadeableIntersections array of shadeable intersections
 * @param pathSegments array of pathsegments
 * @param materials array of materials
 */
__global__ void shadeRealMaterial(
    int iter, int num_paths, ShadeableIntersection *shadeableIntersections, PathSegment *pathSegments, Material *materials, int depth, struct TexData *baseColor, glm::vec3 *backData, int *backWidth, int *backHeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_paths)
        return;
    if (pathSegments[idx].remainingBounces <= 0)
        return;

    ShadeableIntersection intersection = shadeableIntersections[idx];
    if (intersection.t > 0.0f)
    {
        Material material = materials[intersection.materialId];
        // height lines of width pixels ->
        //    u * width, v * height, floor both
        //    nU, nV -> nU + width * nV
        int w = material.texWidth;
        int h = material.texHeight;
        long long tmpidx = uv2Idx(intersection.uvs, w, h);
        if (material.emittance > 0.f) // case thing is light
        {
            glm::vec3 materialColor = material.color;
            pathSegments[idx].color *= (materialColor * material.emittance);
            pathSegments[idx].remainingBounces = 0;
        }
        else if (intersection.useTexture && baseColor[tmpidx].emit) // case texel is emmisive
        {
            pathSegments[idx].color *= 3.f * texCol2Color(baseColor[tmpidx].bCol);
            pathSegments[idx].remainingBounces = 0;
        }
        else // case thing isnt light so calculate
        {
#ifdef DEBUG_SURFACE_NORMAL
            pathSegments[idx].color = intersection.surfaceNormal; // Debug only
            pathSegments[idx].remainingBounces = 0;
#elif defined(DEBUG_T_VAL)
            pathSegments[idx].color = glm::vec3(intersection.t * 0.01); // Debug only
            pathSegments[idx].remainingBounces = 0;
#elif defined(DEBUG_TEX_BASE_COLOR)
            // long long tmpidx = (int)(glm::floor(intersection.uvs.x * w) + w * glm::floor(intersection.uvs.y * h));
            pathSegments[idx].color =
                tmpidx < (w * h) &&
                        intersection.uvs.x >= 0 &&
                        intersection.uvs.y >= 0 &&
                        intersection.useTexture
                    // ? glm::vec3(baseColor[tmpidx].bCol[0] / 255.f, baseColor[tmpidx].bCol[1] / 255.f, baseColor[tmpidx].bCol[2] / 255.f)
                    ? texCol2Color(baseColor[tmpidx].bCol)
                    // : intersection.surfaceNormal;
                    : material.color;
            // pathSegments[idx].color = baseColor[tmpidx] / 255.f;
            //   : material.color; // Debug only
            pathSegments[idx].remainingBounces = 0;
#else
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, depth);
            scatterRay(pathSegments[idx], getPointOnRay(pathSegments[idx].ray, intersection.t), intersection.surfaceNormal, material, rng, baseColor[tmpidx], intersection.useTexture);
            pathSegments[idx].remainingBounces--;
#endif
        }
    }
    else // intersection't => black
    {
        // TODO: Handle case where it goes off to background
        // pathSegments[idx].color = glm::vec3(0.1f);
        // pathSegments[idx].color *= glm::vec3(0.1f);
        // pathSegments[idx].color = glm::vec3(0.f);
        if (backData == NULL)
        {
            pathSegments[idx].color = glm::vec3(0.f);
        }
        else
        {
            glm::vec3 q = 0.5f * pathSegments[idx].ray.direction + 0.5f;
            int index = glm::floor(q.x * (*backWidth)) + (*backWidth) * glm::floor(q.y * (*backHeight));
            pathSegments[idx].color *= backData[index];
        }
        pathSegments[idx].remainingBounces = 0;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3 *image, PathSegment *iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] +=
            iterationPath.color;
        // 0.8f * iterationPath.color + glm::vec3(0.2f);
    }
}

struct orderMaterials
{
    __host__ __device__ bool operator()(ShadeableIntersection const &a, ShadeableIntersection const &b)
    {
        return a.materialId < b.materialId;
    }
};
struct isDeadYet
{
    __host__ __device__ bool operator()(PathSegment const ps)
    {
        return ps.remainingBounces > 0;
    }
};
/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4 *pbo, int frame, int iter)
{
#ifdef TIME_PATHTRACE
    static double timerAcc = 0.0;
#endif
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera &cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, . This has been done
    //   for you.

    // TODO: perform one iteration of path tracing
    int depth = 0;
    // * Initialize array of path rays (using rays that come out of the camera)
    // * You can pass the Camera object to that kernel.
    // * Each path ray must carry at minimum a (ray, color) pair,
    // * where color starts as the multiplicative identity, white = (1, 1, 1).
    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths, depth);
    checkCUDAError("generate camera ray");
    hipDeviceSynchronize();
    checkCUDAError("device synch");

    PathSegment *dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks
#ifdef TIME_PATHTRACE
    using TimerClass = std::chrono::high_resolution_clock::time_point;
    TimerClass start = std::chrono::high_resolution_clock::now();
#endif
#ifdef GROUP_RAYS
    thrust::device_ptr<PathSegment> device_t_paths(dev_paths);
    thrust::device_ptr<ShadeableIntersection> device_t_intersections(dev_intersections);
#endif

    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // cout << "Loop start cpu noncuda" << endl;
        checkCUDAError("startOfLoop");
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));
        // hipDeviceSynchronize();
        checkCUDAError("memset");
        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;

#ifdef CACHE_FIRST
        if (depth == 0) // main increments iteration before calling
        {
            if (iter == 1)
            {
                computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
                    depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersectionCache, dev_tris);
                checkCUDAError("trace first bounce");
                hipDeviceSynchronize();
            }
            hipMemcpy(dev_intersections, dev_intersectionCache, pixelcount * sizeof(ShadeableIntersection), hipMemcpyDeviceToDevice);
            // depth++;
        }
        else
#endif
        {
            computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>>(
                depth, num_paths, dev_paths, dev_geoms, hst_scene->geoms.size(), dev_intersections, dev_tris, dev_texData, dev_materials);
            checkCUDAError("trace one bounce");
            hipDeviceSynchronize();
        }

#ifdef GROUP_RAYS
        thrust::sort_by_key(device_t_intersections, device_t_intersections + num_paths, device_t_paths, orderMaterials());
#endif

        // TODO:
        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        // shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(iter, num_paths, dev_intersections, dev_paths, dev_materials);
        shadeRealMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(iter, num_paths, dev_intersections, dev_paths, dev_materials, depth, dev_texData, dev_background, dev_backWidth, dev_backHeight);
        checkCUDAError("my shader");
        depth++;
#ifdef COMPACT_RAYS
        // Stream compact away all of the terminated paths.
        PathSegment *newEnd = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, isDeadYet());
        num_paths = newEnd - dev_paths;
        if (num_paths < 1)
        {
            depth++;
        }
#endif
        iterationComplete =
            (depth >= traceDepth)
#ifdef COMPACT_RAYS
            || (num_paths <= 0)
#endif
            ; // TODO: should be based off stream compaction results.
    }
    num_paths = dev_path_end - dev_paths;

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(num_paths, dev_image, dev_paths);

#ifdef TIME_PATHTRACE
    hipDeviceSynchronize();
    TimerClass finish = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed(finish - start);
    double fElapsed = static_cast<decltype(fElapsed)>(elapsed.count());
    timerAcc += fElapsed;
#endif
#ifdef TIME_PATHTRACE
    if (iter >= 999)
        std::cout << "elapsed time: " << timerAcc << "miliseconds" << std::endl;
#endif

    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
               pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);
    checkCUDAError("pathtrace");
}
