#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <glm/gtc/matrix_inverse.hpp>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <stb_image.h>
#include "thirdparty/tiny_obj_loader.h"
#include "scene.h"

const dim3 IMAGE_PROCESS_BLOCK_SIZE(16, 16, 1);

__global__ void kernInvGammaCorrect(glm::vec3* dst, stbi_uc* src, int x, int y, int channel) {
    int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    if (idxX < x && idxY < y) {
        int index = Texture2D<glm::vec3>::index2Dto1D(glm::vec2(x, y), idxX, idxY);
        glm::vec3 color;
#pragma unroll
        for (int c = 0; c < channel && c < 3; ++c) {
            stbi_uc byte = src[index * channel + c];
            float comp = byte / 255.f;
            comp = powf(comp, 2.2);
            color[c] = glm::clamp(comp, 0.f, 1.f);
        }
        dst[index] = color;
    }
}


void Scene::addTextureToLoad(size_t id, i64 varOffset, const std::string& filename) {
    auto& pkg = texturePackage;
    auto it = pkg.dstToAddrMap.find(id);
    std::unordered_map<i64, std::string>* map_ptr = nullptr;
    if (it != pkg.dstToAddrMap.end()) {
        map_ptr = &it->second;
    }
    else {
        pkg.dstToAddrMap[id] = std::unordered_map<i64, std::string>();
        map_ptr = &pkg.dstToAddrMap[id];
    }
    (*map_ptr)[varOffset] = filename;
}

Texture2D<glm::vec3> Scene::loadTexture(const std::string& filename) {
    std::cout << "Loading Texture from " << filename << "..." << std::endl;
    auto& pkg = texturePackage;
    auto it = pkg.idMap.find(filename);
    if (it != pkg.idMap.end()) {
        std::cout << "Texture " << filename << " is already exist.\n" << std::endl;
        return pkg.resources[it->second];
    }

    pkg.idMap[filename] = pkg.resources.size();
    pkg.resources.emplace_back();
    auto& res = pkg.resources.back();

    //Texture2D<glm::vec3>& tex = textureBuffers[i];
    int x, y, channel; 
    stbi_set_flip_vertically_on_load(1);
    stbi_uc* imageCPU = stbi_load(filename.c_str(), &x, &y, &channel, 0);
    stbi_uc* imageGPU;
    res.size.x = x;
    res.size.y = y;

    hipMalloc(&imageGPU, sizeof(stbi_uc) * x * y * channel);
    hipMalloc(&res.buffer, sizeof(glm::vec3) * x * y);

    hipMemcpy(imageGPU, imageCPU, sizeof(stbi_uc) * x * y * channel, hipMemcpyHostToDevice);
    dim3 blockCount((x + IMAGE_PROCESS_BLOCK_SIZE.x - 1) / IMAGE_PROCESS_BLOCK_SIZE.x, (y + IMAGE_PROCESS_BLOCK_SIZE.y - 1) / IMAGE_PROCESS_BLOCK_SIZE.y, 1);
    kernInvGammaCorrect<<<blockCount, IMAGE_PROCESS_BLOCK_SIZE>>>(res.buffer, imageGPU, x, y, channel);
    checkCUDAError("kernInvGammaCorrect");

    hipFree(imageGPU);
    hipDeviceSynchronize();
    stbi_image_free(imageCPU);
    checkCUDAError("hipFree imageGPU");
    std::cout << "Texture " << filename << '<' << x << ',' << y << ',' << channel << "> created.\n" << std::endl;

    return res;
}

void Scene::initTextures() {
    for(auto& materialToTexturePair : texturePackage.dstToAddrMap) {
        size_t materialId = materialToTexturePair.first;
        Material& material = materials[materialId];
        for (auto& textureFilePair : materialToTexturePair.second) {
            const std::string& filename = textureFilePair.second;
            Texture2D<glm::vec3>* texture_ptr = utilityCore::getPtrInStruct<Texture2D<glm::vec3>>(&material, textureFilePair.first);
            *texture_ptr = loadTexture(filename);
        }
    }
}

void Scene::freeTextures() {
    for (auto& tex : texturePackage.resources) {
        hipFree(tex.buffer);
    }
    hipDeviceSynchronize();
    checkCUDAError("hipFree textures");
}

void Scene::addModelToLoad(size_t id, i64 varOffset, const std::string& filename) {
    auto& pkg = modelPackage;
    auto it = pkg.dstToAddrMap.find(id);
    std::unordered_map<i64, std::string>* map_ptr = nullptr;
    if (it != pkg.dstToAddrMap.end()) {
        map_ptr = &it->second;
    }
    else {
        pkg.dstToAddrMap[id] = std::unordered_map<i64, std::string>();
        map_ptr = &pkg.dstToAddrMap[id];
    }
    (*map_ptr)[varOffset] = filename;
}

TriMesh Scene::loadModelObj(const std::string& filename) {
    std::cout << "Loading Model from " << filename << "..." << std::endl;
    auto& pkg = modelPackage;
    auto it = pkg.idMap.find(filename);
    if (it != pkg.idMap.end()) {
        std::cout << "Model " << filename << " is already exist.\n" << std::endl;
        return pkg.resources[it->second];
    }

    pkg.idMap[filename] = pkg.resources.size();
    pkg.resources.emplace_back();
    auto& res = pkg.resources.back();

    std::vector<tinyobj::shape_t> shapes;
    std::vector<tinyobj::material_t> materials;
    tinyobj::LoadObj(shapes, materials, filename.c_str());

    res.triangleNum = 0;
    for (tinyobj::shape_t& shape : shapes) {
        res.triangleNum += shape.mesh.indices.size() / 3;
    }

    std::vector<Triangle> tris(res.triangleNum);
    size_t triIdx = 0;
    for (tinyobj::shape_t& shape : shapes) {
        for (int i = 0; i < shape.mesh.indices.size(); i += 3) {
            auto& tri = tris[triIdx];
            tri.triangleid = triIdx;
            tri.twoSided = true;

            //memcpy(tri.position, &shape.mesh.positions[i0 * 3], sizeof(tri.position));
            //memcpy(tri.normal, &shape.mesh.normals[i0 * 3], sizeof(tri.normal));
            //memcpy(tri.uv0, &shape.mesh.texcoords[i0 * 2], sizeof(tri.uv0));

            size_t i0 = shape.mesh.indices[i];
            size_t i1 = shape.mesh.indices[i + 1];
            size_t i2 = shape.mesh.indices[i + 2];

            memcpy(&tri.pos0, &shape.mesh.positions[i0 * 3], sizeof(tri.pos0));
            memcpy(&tri.pos1, &shape.mesh.positions[i1 * 3], sizeof(tri.pos1));
            memcpy(&tri.pos2, &shape.mesh.positions[i2 * 3], sizeof(tri.pos2));

            memcpy(&tri.nrm0, &shape.mesh.normals[i0 * 3], sizeof(tri.nrm0));
            memcpy(&tri.nrm1, &shape.mesh.normals[i1 * 3], sizeof(tri.nrm1));
            memcpy(&tri.nrm2, &shape.mesh.normals[i2 * 3], sizeof(tri.nrm2));

            memcpy(&tri.uv00, &shape.mesh.texcoords[i0 * 2], sizeof(tri.uv00));
            memcpy(&tri.uv01, &shape.mesh.texcoords[i1 * 2], sizeof(tri.uv01));
            memcpy(&tri.uv02, &shape.mesh.texcoords[i2 * 2], sizeof(tri.uv02));
            
            ++triIdx;
        }
    }
    hipMalloc(&res.triangles, sizeof(Triangle) * res.triangleNum);
    hipMemcpy(res.triangles, tris.data(), sizeof(Triangle) * res.triangleNum, hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    checkCUDAError("loadModelObj");
    std::cout << "Model " << filename << '<' << res.triangleNum << "> created.\n" << std::endl;
    return res;
}

void Scene::initModels() {
    for(auto& geomToModelPair : modelPackage.dstToAddrMap) {
        size_t geomId = geomToModelPair.first;
        Geom& geom = geoms[geomId];
        for (auto& modelFilePair : geomToModelPair.second) {
            const std::string& filename = modelFilePair.second;
            TriMesh* model_ptr = utilityCore::getPtrInStruct<TriMesh>(&geom, modelFilePair.first);
            std::string extension = utilityCore::getModelExtension(filename);
            if (stricmp("obj", extension.c_str()) == 0) {
                *model_ptr = loadModelObj(filename);
            }
            //TODO: Other model format.
        }
        //TODO: Build BVH if necessary.
    }
}

void Scene::freeModels() {
    for (auto& mdl : modelPackage.resources) {
        hipFree(mdl.triangles);
    }
    hipDeviceSynchronize();
    checkCUDAError("hipFree models");
}
