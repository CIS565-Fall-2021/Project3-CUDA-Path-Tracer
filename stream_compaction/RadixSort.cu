#include "hip/hip_runtime.h"
#include "RadixSort.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
namespace StreamCompaction {
	namespace RadixSort {
		using StreamCompaction::Common::PerformanceTimer;
		PerformanceTimer& timer()
		{
			static PerformanceTimer timer;
			return timer;
		}

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)  // We can use defines provided in this project

		int* dev_buf;
		int* bufBit;
		int* falseBuf;
		int* trueBuf;
		int* bufNotBits;
		int* bufScatter;
		int* bufAnswer;
#define blockSize 512


		void AllocateMemory(int n)
		{
			hipMalloc((void**)&dev_buf, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_buf failed!");
			hipMalloc((void**)&bufBit, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufloader failed!");
			hipMalloc((void**)&falseBuf, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufB failed!");
			hipMalloc((void**)&trueBuf, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufS failed!");
			hipMalloc((void**)&bufNotBits, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufAnswers failed!");
			hipMalloc((void**)&bufScatter, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufAnswers failed!");
			hipMalloc((void**)&bufAnswer, n * sizeof(int));
			checkCUDAErrorWithLine("hipMalloc dev_bufAnswers failed!");
			hipDeviceSynchronize();
		}

		void FreeMemory() {
			hipFree(dev_buf);
			hipFree(bufBit);
			hipFree(falseBuf);
			hipFree(trueBuf);
			hipFree(bufNotBits);
			hipFree(bufScatter);
			hipFree(bufAnswer);
		}


		__global__ void PopulateBits(int bitOrder, int* bufInputData, int* bufBit, int N)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index > N - 1)
			{
				return;
			}
			int mask = 1 << bitOrder;
			int masked_num = bufInputData[index] & mask;
			int thebit = masked_num >> bitOrder;
			bufBit[index] = thebit;
		}

		__global__ void PopulateNotBits(int *bitNotBits, const int* bufBits, int N)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index > N - 1)
			{
				return;
			}
			if (bufBits[index] == 0)
			{
				bitNotBits[index] = 1;
				return;
			}
			bitNotBits[index] = 0;
		}

		__global__ void CopyAnswerToInputBuf(int* BufAnswer, int* ScatterBuffer, int* InputBuf, int N)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index > N - 1)
			{
				return;
			}
			BufAnswer[ScatterBuffer[index]] = InputBuf[index];
		}


		__global__ void ComputeTArray(int* outputBuf, int *falseBuf, int totalFalses, int N)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index > N - 1)
			{
				return;
			}
			outputBuf[index] = index - falseBuf[index] + totalFalses;
		}

		__global__ void PerformScatter(int* outputBuf, int* inputBuf, int* bitBuf, int*falseBuf, int *trueBuf, int N)
		{
			int index = threadIdx.x + (blockIdx.x * blockDim.x);
			if (index > N - 1)
			{
				return;
			}
			if (bitBuf[index])
			{
				outputBuf[index] = trueBuf[index];
				return;
			}
			outputBuf[index] = falseBuf[index];

		}


		void PerformThrustSort(int n, int* odata, const int* idata)
		{
			thrust::host_vector<int>hstIn(idata, idata + n);
			timer().startGpuTimer();
			// TODO use `thrust::exclusive_scan`
			// example: for device_vectors dv_in and dv_out:
			// thrust::exclusive_scan(dv_in.begin(), dv_in.end(), dv_out.begin());
			thrust::sort(hstIn.begin(), hstIn.end());

			thrust::copy(hstIn.begin(), hstIn.end(), odata);

			timer().endGpuTimer();
		}



		void PerformGPUSort(int n, int* odata, const int* idata)
		{
			AllocateMemory(n);
			hipMemcpy(dev_buf, idata, sizeof(int) * n, hipMemcpyHostToDevice);

			dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
			timer().startGpuTimer();
			for (int i = 0; i < 6; i++)
			{
				PopulateBits << < fullBlocksPerGrid, blockSize >> > (i, dev_buf, bufBit, n);
				hipDeviceSynchronize();
				PopulateNotBits << < fullBlocksPerGrid, blockSize >> > (bufNotBits, bufBit, n);
				hipDeviceSynchronize();

				int* inputNotBits= new int[n];
				hipMemcpy(inputNotBits, bufNotBits, n * sizeof(int), hipMemcpyDeviceToHost);
				Efficient::scan(n, odata, inputNotBits);
				hipMemcpy(falseBuf, odata, n * sizeof(int), hipMemcpyHostToDevice);

				int TotalFalses = inputNotBits[n - 1] + odata[n - 1];
				ComputeTArray << < fullBlocksPerGrid, blockSize >> > (trueBuf, falseBuf, TotalFalses, n);
				hipDeviceSynchronize();
				PerformScatter << < fullBlocksPerGrid, blockSize >> > (bufScatter, dev_buf, bufBit, falseBuf, trueBuf, n);
				hipDeviceSynchronize();
				CopyAnswerToInputBuf << < fullBlocksPerGrid, blockSize >> > (bufAnswer, bufScatter,  dev_buf, n);
				hipDeviceSynchronize();
				std::swap(dev_buf, bufAnswer);
				hipDeviceSynchronize();
			}
			timer().endGpuTimer();
			hipMemcpy(odata, dev_buf, sizeof(int) * n, hipMemcpyDeviceToHost);
		}

	}
}