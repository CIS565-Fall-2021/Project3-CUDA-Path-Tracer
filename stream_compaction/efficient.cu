#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"


namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }


        // only run for 1 thread at a time
        __global__ void kernUpdateArr(int idx, int val, int *arr) {
            arr[idx] = val;
        }

        __global__ void kernScanUpSweep(int n, int *data, int pow2) {
            /*int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= n) {
                return;
            }

            if (idx % (2 * pow2) == 0) {
                data[idx + 2 * pow2 - 1] += data[idx + pow2 - 1];
            }*/

            // optimized solution
            size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            idx = 2 * pow2 * (idx + 1) - 1;
            if (idx >= n) {
                return;
            }
            data[idx] += data[idx - pow2];
            
        }

        __global__ void kernScanDownSweep(int n, int *data, int pow2) {
            /*int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (idx >= n) {
                return;
            }

            if (idx % (2 * pow2) == 0) {
                int temp = data[idx + pow2 - 1];
                data[idx + pow2 - 1] = data[idx + 2 * pow2 - 1];
                data[idx + 2 * pow2 - 1] += temp;
            }*/
            // optimized solution
            size_t idx = (blockIdx.x * blockDim.x) + threadIdx.x;
            idx = 2 * pow2 * (idx + 1) - 1;
            if (idx >= n) {
                return;
            }
            int temp = data[idx - pow2];
            data[idx - pow2] = data[idx];
            data[idx] += temp;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            int* dev_arr;
            int maxDepth = ilog2ceil(n);
            int size = pow(2, maxDepth);

            hipMalloc((void**)&dev_arr, size * sizeof(int));
            hipMemcpy(dev_arr, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 blockNumPow((size + blockSize - 1) / blockSize);


            timer().startGpuTimer();
            for (int d = 0; d < maxDepth; d++) {
                blockNumPow = (size / pow(2, d + 1) + blockSize - 1) / blockSize;
                kernScanUpSweep << <blockNumPow, blockSize >> > (size, dev_arr, pow(2, d));
            }

            kernUpdateArr << <1, 1 >> > (size - 1, 0, dev_arr);

            for (int d = maxDepth - 1; d >= 0; d--) {
                blockNumPow = (size / pow(2, d + 1) + blockSize - 1) / blockSize;
                kernScanDownSweep << <blockNumPow, blockSize >> > (size, dev_arr, pow(2, d));
            }
            timer().endGpuTimer();
            hipMemcpy(odata, dev_arr, n * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(dev_arr);
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int *odata, const int *idata) {
            int* dev_idata, *dev_odata, *dev_bools, *dev_indices;
            int maxDepth = ilog2ceil(n);
            int size = pow(2, maxDepth);
   
            hipMalloc((void**)&dev_idata, size * sizeof(int));
            hipMalloc((void**)&dev_odata, n * sizeof(int));
            hipMalloc((void**)&dev_bools, size * sizeof(int));
            hipMalloc((void**)&dev_indices, size * sizeof(int));
            hipMemcpy(dev_idata, idata, n * sizeof(int), hipMemcpyHostToDevice);

            dim3 blockNum((n + blockSize - 1) / blockSize);
            dim3 blockNumPow((size + blockSize - 1) / blockSize);

            timer().startGpuTimer();
            Common::kernMapToBoolean << <blockNum, blockSize >> > (size, dev_bools, dev_idata);
            hipMemcpy(dev_indices, dev_bools, size * sizeof(int), hipMemcpyHostToDevice);

            // scan
            for (int d = 0; d < maxDepth; d++) {
                blockNumPow = (size / pow(2, d + 1) + blockSize - 1) / blockSize;
                kernScanUpSweep << <blockNumPow, blockSize >> > (size, dev_indices, pow(2, d));
            }

            kernUpdateArr << <1, 1 >> > (size - 1, 0, dev_indices);

            for (int d = maxDepth - 1; d >= 0; d--) {
                blockNumPow = (size / pow(2, d + 1) + blockSize - 1) / blockSize;
                kernScanDownSweep << <blockNumPow, blockSize >> > (size, dev_indices, pow(2, d));
            }

            // scatter
            Common::kernScatter << <blockNum, blockSize >> > (n, dev_odata, dev_idata, dev_bools, dev_indices);
            timer().endGpuTimer();

            hipMemcpy(odata, dev_odata, n * sizeof(int), hipMemcpyDeviceToHost);
            int* bools = new int[n];
            hipMemcpy(bools, dev_bools, n * sizeof(int), hipMemcpyDeviceToHost);
            int count = 0;
            for (int i = 0; i < n; i++) {
                if (bools[i]) {
                    count++;
                }
            }

            hipFree(dev_idata);
            hipFree(dev_odata);
            hipFree(dev_bools);
            hipFree(dev_indices);
            return count;
        }
    }
}
